﻿

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <chrono>
#define size 1024
int main()
{
	auto begin = std::chrono::steady_clock::now();
	FILE* S1;
	S1 = fopen("RESULT.txt", "w");
	int i, j = 0, r, k = 0, f = 0, l = 0;
	float sum = 0;
	float** a;
	float** b;
	a = new float* [size];
	b = new float* [size];
	for (i = 0; i < size; i++)
	{
		a[i] = new float[size];
		b[i] = new float[size];
	}
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			r = rand() % 2;
			if (r == 0)
				a[i][j] = 0;
			else
				a[i][j] = rand()%8;
			b[i][j] = 1;
			//printf("%0.0f", a[i][j]);
		}
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			if ((a[i][j - 1] == 0 && a[i][j + 1] == 0) || a[i][j] == 0/*||a[i][j]<s/n*/)
			{
				b[i][j] = 0;
			}
		}
	for (i = 0; i < size; i++)
	{
		for (j = 0; j < size; j++)
			fprintf(S1, "%0.0f ", b[i][j]);
		fprintf(S1, "\n");
	}
	for (i = 0; i < size; i++)
	{
		for (j = 0; j < size ; j++)
		{
			if (b[i][j] != 0)
				sum += a[i][j];
		}
		a[i][0] = sum;
		sum = 0;
	}
	for (i = 0; i < size; i++)
		{
		fprintf(S1, "%0.0f\n", a[i][0]);
		}
	auto end = std::chrono::steady_clock::now();
	auto elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin);
	std::cout << "The time: " << elapsed_ms.count() << " ms\n";
}
