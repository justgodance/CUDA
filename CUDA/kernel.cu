#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <chrono>
#ifndef __HIPCC__  
#define __HIPCC__
#endif
#define NSIZE 8
__global__ void Mass(float* dA, float* dB, int size)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y; // определяем индексы нитей и блоков
	float res = 0;
	for (int k = 0; k < NSIZE-1; k++)
		if (dA[NSIZE * (blockDim.y * blockIdx.y + threadIdx.y) + k] != 0 && dA[NSIZE * (blockDim.y * blockIdx.y + threadIdx.y) + k + 1] != 0)
		{
			dB[NSIZE * (blockDim.y * blockIdx.y + threadIdx.y) + k] = 1;
				dB[NSIZE * (blockDim.y * blockIdx.y + threadIdx.y) + k + 1] = 1; //ищем пятна
		}
	for (int k = 0; k < NSIZE; k++)
	{
		if (dB[NSIZE * (blockDim.y * blockIdx.y + threadIdx.y) + k] != 0)
		res += dA[NSIZE * (blockDim.y * blockIdx.y + threadIdx.y) + k]; // считаем интенсивность
	}
	dA[NSIZE * (blockDim.y * blockIdx.y + threadIdx.y)]= res; // записываем интенсивность
}
int main()
{
	FILE* S1;
	S1 = fopen("RESULT.txt", "w");
	int i, j = 0, r, k = 0, f = 0, l = 0;
	float* hA, * hB; //hA - массив изначальных данных;hB - массив по поиску пятен
	float timerValueGPU, timerValueCPU; // измеряем время
	hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	hipEventRecord(start, 0);
	size_t size = sizeof(float) * NSIZE * NSIZE; //общий размер
	hA = (float*)malloc(size); 
	hB = (float*)malloc(size); // определяем массивы для CPU
	for (i = 0; i < NSIZE; i++) // цикл по заполнению массива данными
	{
		for (j = 0; j < NSIZE; j++)
		{
			r = rand() % 2;
			if (r == 0)
			{
				hA[j + i * NSIZE] = 0;
				printf("%0.0f  ", hA[j + i * NSIZE]);
			}
			else
			{
				hA[j + i * NSIZE] = rand()%8;
				printf("%0.0f  ", hA[j + i * NSIZE]);
			}
		}
	printf("\n");
	}
	float* dA = NULL;
	float* dB = NULL; 
	hipMalloc((void**)&dA, size);
	hipMalloc((void**)&dB, size); // определяем массивы для GPU 
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice); // копируем массив данных на GPU
	dim3 threads(NSIZE, 1);
	dim3 blocks(1, NSIZE); // определяем размер блоков
	printf("\n");
	Mass <<< blocks, threads >>> (dA, dB, size); // переходим в функцию ядро
	hipMemcpy(hA, dA, size,hipMemcpyDeviceToHost);
	hipMemcpy(hB, dB, size, hipMemcpyDeviceToHost); // копируем из GPU в CPU
	fprintf(S1, "INTENSITY \n");
	for (i = 0; i < NSIZE; i++)
	{
		fprintf(S1, "%0.0f  ", hA[i * NSIZE]);
		fprintf(S1, "\n");
	}
	fprintf(S1, "\n");
	for (i = 0; i < NSIZE; i++)
	{
		for (j = 0; j < NSIZE; j++)
			fprintf(S1, "%0.0f  ", hB[i * NSIZE+j]);
		fprintf(S1, "\n");
	}												  // заполняем файл данными
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueGPU, start, stop);
	printf("\n GPU calculation time %f msec\n", timerValueGPU); // проверяем время
	hipFree(dB);									    
	hipFree(dA);
	free(hA);
	free(hB); // освобождаем память
}


